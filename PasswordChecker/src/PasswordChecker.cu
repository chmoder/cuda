/*
 ============================================================================
 Name        : PasswordChecker.cu
 Author      : Thomas Cross
 Version     : 0.0.1
 Copyright   : This is free.  Do with it as you please
 Description : CUDA password generator and checker
 ============================================================================
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <cmath>

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/*
 * The password in question
 * */
__constant__ char password[16];
__constant__ char alphabet[95];

/**
 * CUDA kernel copies one string buffer to another
 */
__device__ char *strcpyDevice(char *dest, const char *src)
{
    char *ret = dest;
    while (*dest++ = *src++)
        ;
    return ret;
}

/**
 * CUDA kernel that compares two strings
 */
__device__ int strcmpDevice(const char * s1, const char * s2)
{
	while(*s1 && (*s1==*s2))
	{
		s1++,s2++;
	}
	return *(const unsigned char*)s1-*(const unsigned char*)s2;
}

/**
 * CUDA kernel that computes converts base 10 to any base
 * found this online somewhere
 */
__device__ void convertBase(char converted_string[], int converted_number[], unsigned long long number_to_convert, int base, char *alphabet) {
	//char alphabet[95] = {'0', '1', '2', '3', '4', '5', '6', '7', '8', '9', 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z', 'A', 'B', 'C', 'D', 'E', 'F', 'G', 'H', 'I', 'J', 'K', 'L', 'M', 'N', 'O', 'P', 'Q', 'R', 'S', 'T', 'U', 'V', 'W', 'X', 'Y', 'Z', '!', '"', '#', '$', '%', '&', "'", '(', ')', '*', '+', ',', '-', '.', '/', ':', ';', '<', '=', '>', '?', '@', '[', '\\', ']', '^', '_', '`', '{', '|', '}', '~', ' '};
	//char *alphabet = "0123456789abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ!\"#$%&\'()*+,-./:;<=>?@[\\]^_`{|}~ ";
	int index = 0;
	base = 95;

	/* convert to the indicated base */
	while (number_to_convert != 0)
	{
		converted_number[index] = number_to_convert % base;
		number_to_convert = number_to_convert / base;
		++index;
	}
	converted_string[index] = '\0';

	/* now print the result in reverse order */
	--index;  /* back up to last entry in the array */
	int word_length = index;
	for(  ; index>=0; index--) /* go backward through array */
	{
		converted_string[word_length - index] = alphabet[converted_number[index]];
	}
}

__global__ void checkPasswordShared(char *return_guess, const int string_size, const int iteration) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int total_threads = blockDim.x * gridDim.x;
	int converted_number[16];
	char converted_string[16];
	unsigned long long codex = idx + (total_threads * iteration);
	unsigned long long codex_for_printf = idx + (total_threads * iteration);
	const int base = (int)'z';

    convertBase(converted_string, converted_number, codex, base, alphabet);

	if(strcmpDevice(converted_string, password) == 0)
	{
		printf("%llu,%d,%d,%d,%d,%d, %s == %s\n", codex_for_printf, blockIdx.x, blockDim.x, threadIdx.x, total_threads, iteration, converted_string, password);
		return_guess = strcpyDevice(return_guess, converted_string);
	}
}

/**
 * Host function that copies the data and launches the work on GPU
 * Created n streams where n = number of multiprocessors * 8 (peformance degrades after this point on my GTX)
 * thread count per kernel is your max threads / 2
 * block count is the number of multiprocessors you have
 * Using shared memory and registers I have been measuring about 32,499,876 password generations and comparisons per second
 */
char *checkPasswordHost(int iteration)
{
	hipSetDevice(0);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	int STREAM_COUNT = deviceProp.multiProcessorCount * 8 * 8;
	hipStream_t streams[STREAM_COUNT];

	for(int i = 0; i < STREAM_COUNT; ++i)
	{
		hipStreamCreate(&streams[i]);
	}

	static const int THREAD_COUNT = deviceProp.maxThreadsPerMultiProcessor / 2;
	static const int BLOCK_COUNT = deviceProp.multiProcessorCount;
	static const int SIZE = 16;
	char *converted_string = new char[SIZE];
	char *gpuData;

	for(int i = 0; i < SIZE; ++i)
		converted_string[i] = '\0';

	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuData, sizeof(char)*SIZE));
	CUDA_CHECK_RETURN(hipMemcpy(gpuData, converted_string, sizeof(char)*SIZE, hipMemcpyHostToDevice));

	for(int i = 0; i < STREAM_COUNT; ++i)
	{
		checkPasswordShared<<<BLOCK_COUNT, THREAD_COUNT, 0, streams[i]>>> (gpuData, SIZE, (iteration * STREAM_COUNT) + i);
	}

	for(int i = 0; i < STREAM_COUNT; ++i)
	{
		hipStreamSynchronize(streams[i]);
		hipStreamDestroy(streams[i]);
	}

	CUDA_CHECK_RETURN(hipMemcpy(converted_string, gpuData, sizeof(char)*SIZE, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(gpuData));
	return converted_string;
}

int main(void)
{
    int iteration = 0;
    int max_iterations = 100000000;
    char *answer_password;
    answer_password = new char[1];
    answer_password[0] = '\0';
    std::string temp_password;

	std::cout << "Please enter a password to find: ";
	getline(std::cin, temp_password);
	CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(password), temp_password.c_str(), sizeof(char) * 16));
	CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(alphabet), "0123456789abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ!\"#$%&\'()*+,-./:;<=>?@[\\]^_`{|}~ ", sizeof(char) * 95));
	std::cout << "searching for \"" << temp_password.c_str() << "\"..." << std::endl;

	time_t start = time(0);
    while(answer_password[0] == '\0' && iteration < max_iterations)
	{
		delete[] answer_password;
		answer_password = checkPasswordHost(iteration);
    	//std::cout << "The password could be: \"" << answer_password << "\"" << std::endl;
		iteration++;
	}

    if(answer_password[0] != '\0')
    {
    	std::cout << "The password is: \"" << answer_password << "\"" << std::endl;
    }
    else if(iteration == max_iterations)
    {
    	std::cout << "Reached max iterations of " << max_iterations << std::endl;
    }

	time_t end = time(0);
	double time = difftime(end, start);
	std::cout << "Execution Time: " << (int)floor(time) << " seconds" << std::endl;

	delete[] answer_password;

	return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}
