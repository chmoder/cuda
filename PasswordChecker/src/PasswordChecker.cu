/*
 ============================================================================
 Name        : PasswordChecker.cu
 Author      : Thomas Cross
 Version     : 0.0.1
 Copyright   : This is free.  Do with it as you please
 Description : CUDA password generator and checker
 ============================================================================
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <cmath>

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/*
 * The password in question
 * */
__constant__ char password[5] = "Erin";


/**
 * CUDA kernel copies one string buffer to another
 */
__device__ char *strcpyDevice(char *dest, const char *src)
{
    char *ret = dest;
    while (*dest++ = *src++)
        ;
    return ret;
}

/**
 * CUDA kernel that compares two strings
 */
__device__ int strcmpDevice(const char * s1, const char * s2)
{
	while(*s1 && (*s1==*s2))
	{
		s1++,s2++;
	}
	return *(const unsigned char*)s1-*(const unsigned char*)s2;
}

/**
 * CUDA kernel that computes converts base 10 to any base
 * found this online somewhere
 */
__device__ char *convertBase(long number_to_convert, int base) {
	   __shared__ int converted_number[8];
	   //char *converted_string = new char[8];
	   __shared__ char converted_string[8];
	   int index = 0;

	   /* convert to the indicated base */
	   while (number_to_convert != 0)
	   {
	         converted_number[index] = number_to_convert % base;
	         number_to_convert = number_to_convert / base;
	         ++index;
	   }
	   converted_string[index] = '\0';

	   /* now print the result in reverse order */
	   --index;  /* back up to last entry in the array */
	   int word_length = index;
	   for(  ; index>=0; index--) /* go backward through array */
	   {
	         converted_string[word_length - index] = converted_number[index]+(int)' ';
	   }

	   return converted_string;
}

/**
 * CUDA kernel that computes converts base 10 to any base
 * found this online somewhere
 */
__device__ void convertBase(char converted_string[], int converted_number[], int number_to_convert, int base) {
	   int index = 0;

	   /* convert to the indicated base */
	   while (number_to_convert != 0)
	   {
	         converted_number[index] = number_to_convert % base;
	         number_to_convert = number_to_convert / base;
	         ++index;
	   }
	   converted_string[index] = '\0';

	   /* now print the result in reverse order */
	   --index;  /* back up to last entry in the array */
	   int word_length = index;
	   for(  ; index>=0; index--) /* go backward through array */
	   {
	         converted_string[word_length - index] = converted_number[index]+(int)' ';
	   }
}

__global__ void universalCheckPasswordShared(char *return_guess, const int string_size, const int iteration) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int total_threads = blockDim.x * gridDim.x;
	int codex = idx + (total_threads * iteration);
	int codex_for_printf = idx + (total_threads * iteration);
	const int base = (int)'~'+1;

	int converted_number[8];
	char converted_string[8];

    convertBase(converted_string, converted_number, codex, base);

	if(strcmpDevice(converted_string, password) == 0)
	{
		printf("%d,%d,%d,%d,%d,%d, %s == %s\n", codex_for_printf, blockIdx.x, blockDim.x, threadIdx.x, total_threads, iteration, converted_string, password);
		return_guess = strcpyDevice(return_guess, converted_string);
	}


}

/**
 * Host function that copies the data and launches the work on GPU
 * Created n streams where n = number of multiprocessors * 8 (peformance degrades after this point on my GTX)
 * thread count per kernel is your max threads / 2
 * block count is the number of multiprocessors you have
 * Using shared register memory I have been measuring about 32,499,876 password generations and comparisons per second
 */
char *checkPasswordHost(int iteration)
{
	hipSetDevice(0);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	int STREAM_COUNT = deviceProp.multiProcessorCount * 8;
	hipStream_t streams[STREAM_COUNT];

	for(int i = 0; i < STREAM_COUNT; ++i)
	{
		hipStreamCreate(&streams[i]);
	}

	static const int THREAD_COUNT = deviceProp.maxThreadsPerMultiProcessor / 2;
	static const int BLOCK_COUNT = deviceProp.multiProcessorCount;
	//static const int THREAD_COUNT = 1024;
	//static const int BLOCK_COUNT = 16;
	static const int TOTAL_THREADS = THREAD_COUNT * BLOCK_COUNT;
	static const int SIZE = 8;
	char *converted_string = new char[SIZE];
	char **converted_strings;
	int **converted_numbers;
	// This is the variable that the data will be reutrned to.  It is shared amongst all the threads and streams.
	char *gpuData;

	for(int i = 0; i < SIZE; ++i)
		converted_string[i] = '\0';

	CUDA_CHECK_RETURN(hipMalloc((void **)&converted_strings, sizeof(char*)*TOTAL_THREADS));
	CUDA_CHECK_RETURN(hipMalloc((void **)&converted_numbers, sizeof(int*)*TOTAL_THREADS));
	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuData, sizeof(char)*SIZE));
	CUDA_CHECK_RETURN(hipMemcpy(gpuData, converted_string, sizeof(char)*SIZE, hipMemcpyHostToDevice));

	for(int i = 0; i < STREAM_COUNT; ++i)
	{
		universalCheckPasswordShared<<<BLOCK_COUNT, THREAD_COUNT, 0, streams[i]>>> (gpuData, SIZE, (iteration * STREAM_COUNT) + i);
	}

	for(int i = 0; i < STREAM_COUNT; ++i)
	{
		hipStreamSynchronize(streams[i]);
		hipStreamDestroy(streams[i]);
	}

	CUDA_CHECK_RETURN(hipMemcpy(converted_string, gpuData, sizeof(char)*SIZE, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(gpuData));
	CUDA_CHECK_RETURN(hipFree(converted_strings));
	CUDA_CHECK_RETURN(hipFree(converted_numbers));
	return converted_string;
}

int main(void)
{
	time_t start = time(0);
    int iteration = 0;
    int max_iterations = 1000000;
    char *answer_password;
    answer_password = new char[1];
    answer_password[0] = '\0';

    while(answer_password[0] == '\0' && iteration < max_iterations)
	{
		delete[] answer_password;
		answer_password = checkPasswordHost(iteration);
    	//std::cout << "The password could be: \"" << answer_password << "\"" << std::endl;
		iteration++;
	}

    if(answer_password[0] != '\0')
    {
    	std::cout << "The password is: \"" << answer_password << "\"" << std::endl;
    }
    else if(iteration == max_iterations)
    {
    	std::cout << "Reached max iterations of " << max_iterations << std::endl;
    }

	time_t end = time(0);
	double time = difftime(end, start);
	std::cout << "Execution Time: " << (int)floor(time) << " seconds" << std::endl;

	delete[] answer_password;

	return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}

